#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
#include "memory_allocation.h"

__global__ void add(int *d_a, int *d_b, int *h_c, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        h_c[i] = d_a[i] + d_b[i];
    }
}

__global__ void sub(int *d_a, int *d_b, int  *h_c, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        h_c[i] = d_a[i] - d_b[i];
    }
}

__global__ void mult(int *d_a, int *d_b, int  *h_c, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        h_c[i] = d_a[i] * d_b[i];
    }
}

__global__ void mod(int *d_a, int *d_b, int  *h_c, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        h_c[i] = d_a[i] % d_b[i];
    }
}

__host__ std::tuple<int *, int *> allocateRandomHostMemory(int numElements)
{
    srand(time(0));
    size_t size = numElements * sizeof(int);

    // Allocate the host input vector A
    // FILL IN HOST AND DEVICE MEMORY ALLOCATION CODE - SPECIFICALLY h_a needs to be pageable memory
    int *h_a = (int *)malloc(size); /// NOTE: the diff b/w this and hipMalloc syntax
    // Verify that allocations succeeded
    if (h_a == NULL)
    {
        fprintf(stderr, "Failed to allocate host vector h_a!\n");
        exit(EXIT_FAILURE);
    }

    // Allocate the host pinned memory input pointer B
    // FILL IN HOST AND DEVICE MEMORY ALLOCATION CODE - SPECIFICALLY h_b needs to be pinned memory
    /// NOTE: I don't think the pinned feature will be reflected on host side code - just on device side.
    int *h_b = (int *)malloc(size);

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_a[i] = rand() % 100;
        h_b[i] = rand() % 100;
    }

    return {h_a, h_b};
}

// Based heavily on https://www.gormanalysis.com/blog/reading-and-writing-csv-files-with-cpp/
// Presumes that there is no header in the csv file
__host__ std::tuple<int *, int *, int>readCsv(std::string filename)
{
    vector<int> tempResult;
    // Create an input filestream
    ifstream myFile(filename);

    // Make sure the file is open
    if(!myFile.is_open()) throw runtime_error("Could not open file");

    // Helper vars
    string line, colname;
    int val;

    // Read 1st line of data
    getline(myFile, line);
    // Create a stringstream of the current line
    stringstream ss0(line);
    
    // Extract each integer
    while(ss0 >> val){
        tempResult.push_back(val);
        // If the next token is a comma, ignore it and move on
        if(ss0.peek() == ',') ss0.ignore();
    }

    int numElements = tempResult.size();
    // Allocate the host input vector A
    int *h_a = (int *)malloc(numElements*sizeof(int));
    // Copy all elements of vector to input_a
    copy(tempResult.begin(), tempResult.end(), h_a);
    tempResult.clear();

    // Read 2nd line of data
    getline(myFile, line);
    // Create a stringstream of the current line
    stringstream ss1(line);
    
    // Extract each integer
    while(ss1 >> val){
        tempResult.push_back(val);
        // If the next token is a comma, ignore it and move on
        if(ss1.peek() == ',') ss1.ignore();
    }

    // Allocate the host pinned memory input pointer B
    int *h_b;
    hipHostMalloc((int **)&h_b, numElements*sizeof(int));

    // Copy all elements of vector to input_a
    copy(tempResult.begin(), tempResult.end(), h_b);

    // Close file
    myFile.close();
    return {h_a, h_b, numElements};
}

__host__ std::tuple<int *, int *> allocateDeviceMemory(int numElements)
{
    // Allocate the device input vector A
    int *d_a = NULL;
    size_t size = numElements * sizeof(int);
    hipError_t err;
    // (DONE) FILL IN HOST AND DEVICE MEMORY ALLOCATION CODE - SPECIFICALLY h_a needs to handle pageable memory and copies
    /// NOTE: Understand this as: allocate device memory of size=size and assign to ptr d_a
    err = hipMalloc(&d_a, size); 

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_a (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int *d_b;
    // (DONE) FILL IN HOST AND DEVICE MEMORY ALLOCATION CODE - SPECIFICALLY h_b needs to handle pinned memory and copies
    err = hipHostMalloc(&d_b, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_a (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    return {d_a, d_b};
}

__host__ void copyFromHostToDevice(int *h_a, int *h_b, int *d_a, int *d_b, int numElements)
{
    size_t size = numElements * sizeof(int);

    hipError_t err = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy constant int d_v from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__host__ void executeKernel(int *d_a, int *d_b, int *h_c, int numElements, int threadsPerBlock, std::string mathematicalOperation)
{
    // Launch the search CUDA Kernel
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    if (!strcmp(mathematicalOperation.c_str(), "sub"))
    {
        sub<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, h_c, numElements);
    } else if (!strcmp(mathematicalOperation.c_str(), "mult"))
    {
        mult<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, h_c, numElements);
    } else if (!strcmp(mathematicalOperation.c_str(), "mod"))
    {
        mod<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, h_c, numElements);
    } else {
        add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, h_c, numElements);
    }
    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Free device global memory
__host__ void deallocateMemory(int *d_a, int *d_b)
{

    hipError_t err = hipFree(d_a);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_a (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipHostFree(d_b);
    if (err != hipSuccess)
    {
        fprintf(stderr, "[227] Failed to free device vector d_b (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

}

// Reset the device and exit
__host__ void cleanUpDevice()
{
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipError_t err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__host__ void outputToFile(std::string currentPartId, int *h_a, int *h_b, int *h_c, int numElements, std::string mathematicalOperation)
{
	string outputFileName = "output-" + currentPartId + ".txt";
	// NOTE: Do not remove this output to file statement as it is used to grade assignment,
	// so it should be called by each thread
	ofstream outputFile;
	outputFile.open (outputFileName, ofstream::app);

    outputFile << "Mathematical Operation: " << mathematicalOperation << "\n";
    outputFile << "PartID: " << currentPartId << "\n";
    outputFile << "Input A: ";
    for (int i = 0; i < numElements; ++i)
        outputFile << h_a[i] << " ";
    outputFile << "\n";
    outputFile << "Input B: ";
    for (int i = 0; i < numElements; ++i)
        outputFile << h_b[i] << " ";
    outputFile << "\n";
    outputFile << "Result: ";
    for (int i = 0; i < numElements; ++i)
        outputFile << h_c[i] << " ";
    outputFile << "\n";

	outputFile.close();
}

__host__ std::tuple<int, std::string, int, std::string, std::string> parseCommandLineArguments(int argc, char *argv[])
{
    int numElements = 10;
    int threadsPerBlock = 256;
    std::string currentPartId = "test";
    std::string mathematicalOperation = "add";
    std::string inputFilename = "NULL";

    for(int i = 1; i < argc; i++)
    {
        std::string option(argv[i]);
        i++;
        std::string value(argv[i]);
        if(option.compare("-t") == 0) 
        {
            threadsPerBlock = atoi(value.c_str());
        }
        else if(option.compare("-n") == 0) 
        {
            numElements = atoi(value.c_str());
        }
        else if(option.compare("-f") == 0) 
        {
            inputFilename = value;
        }
        else if(option.compare("-p") == 0) 
        {
            currentPartId = value;
        }
        else if(option.compare("-o") == 0) 
        {
            mathematicalOperation = value;
        }
    }

    return {numElements, currentPartId, threadsPerBlock, inputFilename, mathematicalOperation};
}

__host__ std::tuple<int *, int *, int> setUpInput(std::string inputFilename, int numElements)
{
    srand(time(0));
    int *h_a;
    int *h_b;

    if(inputFilename.compare("NULL") != 0)
    {
        tuple<int *, int*, int>csvData = readCsv(inputFilename);
        h_a = get<0>(csvData);
        h_b = get<1>(csvData);
        numElements = get<2>(csvData);
    }
    else 
    {
        tuple<int *, int*> randomData = allocateRandomHostMemory(numElements);
        h_a = get<0>(randomData);
        h_b = get<1>(randomData);
    }

    return {h_a, h_b, numElements};
}

/*
 * Host main routine
 * -n numElements - the number of elements of random data to create
 * -f inputFile - the file for non-random input data
 * -o mathematicalOperation - this will decide which math operation kernel will be executed
 * -p currentPartId - the Coursera Part ID
 * -t threadsPerBlock - the number of threads to schedule for concurrent processing
 */
int main(int argc, char *argv[])
{
    auto[numElements, currentPartId, threadsPerBlock, inputFilename, mathematicalOperation] = parseCommandLineArguments(argc, argv);
    tuple<int *, int*, int> searchInputTuple = setUpInput(inputFilename, numElements);
    int *h_a;
    int *h_b;

    h_a = get<0>(searchInputTuple);
    h_b = get<1>(searchInputTuple);
    numElements = get<2>(searchInputTuple);

    // FILL IN HOST AND DEVICE MEMORY ALLOCATION CODE - SPECIFICALLY h_c needs to be unified memory
    int size = numElements * sizeof(int);
    // int *h_c = (float *)malloc(size);
    int *h_c;
    /// NOTE: Note how we define only h_c and never d_c and also populate this variable in the kernel. 
    /// This saves code space but is slow (as its unified)
    hipError_t err = hipMallocManaged((void**)&h_c, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector h_c (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    auto[d_a, d_b] = allocateDeviceMemory(numElements);
    copyFromHostToDevice(h_a, h_b, d_a, d_b, numElements);

    executeKernel(d_a, d_b, h_c, numElements, threadsPerBlock, mathematicalOperation);

    outputToFile(currentPartId, h_a, h_b, h_c, numElements, mathematicalOperation);

    deallocateMemory(d_a, d_b);

    cleanUpDevice();
    return 0;
}