#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
#include "simpleLinearBlurFilter.hpp"

/*
 * CUDA Kernel Device code
 *
 */
__global__ void applySimpleLinearBlurFilter(uchar *r, uchar *g, uchar *b)
{
    // Not using the 2d grid + shared memory approach

    /* 
    Using the CPU version as reference... as both versions are to have min mean diff

    for(int y = 0; y < rows; ++y)
    {
        for(int x = 1; x < columns-1; ++x)
        {
            Vec3b rgb0 = img.at<Vec3b>(y, x-1);
            Vec3b rgb1 = img.at<Vec3b>(y, x);
            Vec3b rgb2 = img.at<Vec3b>(y, x+1);
            b[y*rows+x] = (rgb0[0] + rgb1[0] + rgb2[0])/3;
            g[y*rows+x] = (rgb0[1] + rgb1[1] + rgb2[1])/3;
            r[y*rows+x] = (rgb0[2] + rgb1[2] + rgb2[2])/3;
        }
    }
    */

    int threadId = blockDim.x * blockIdx.x + threadIdx.x; // because we have a 1d grid and 1d block for 2d - serialised img
    int max_pixels = d_rows * d_columns;
    if(threadId < max_pixels) // total_threads = d_rows * d_columns
    {
        // Not using the 2d grid + shared memory approach

        // Apply a simple filter that averages the RGB values to the left and right of the pixel at the current thread id location
        // Another area for improvement is handling when the current thread is at the let or right edge of the imput image
        /*
            Edge case: as our blur 1d kernel takes the i-1 , i and i+1 for ith pixel, we cant run it for the first and last column. 
            So as per the CPU version, we ignore them. 
        */
        int row = threadId / d_columns;
        int col = threadId % d_columns;

        if(col == 0 || col == d_columns-1) return;

        // for all remaining indices, just take avg 
        r[threadId] = (r[threadId-1]+r[threadId]+r[threadId+1]) / 3;
        g[threadId] = (g[threadId-1]+g[threadId]+g[threadId+1]) / 3;
        b[threadId] = (b[threadId-1]+b[threadId]+b[threadId+1]) / 3;
    }

}

__host__ float compareColorImages(uchar *r0, uchar *g0, uchar *b0, uchar *r1, uchar *g1, uchar *b1, int rows, int columns)
{
    cout << "Comparing actual and test pixel arrays\n";
    int numImagePixels = rows * columns;
    int imagePixelDifference = 0.0;

    for(int r = 0; r < rows; ++r)
    {
        for(int c = 0; c < columns; ++c)
        {
            uchar image0R = r0[r*rows+c];
            uchar image0G = g0[r*rows+c];
            uchar image0B = b0[r*rows+c];
            uchar image1R = r1[r*rows+c];
            uchar image1G = g1[r*rows+c];
            uchar image1B = b1[r*rows+c];
            imagePixelDifference += ((abs(image0R - image1R) + abs(image0G - image1G) + abs(image0B - image1B))/3);
        }
    }

    float meanImagePixelDifference = imagePixelDifference / numImagePixels;
    float scaledMeanDifferencePercentage = (meanImagePixelDifference / 255);
    printf("meanImagePixelDifference: %f scaledMeanDifferencePercentage: %f\n", meanImagePixelDifference, scaledMeanDifferencePercentage);
    return scaledMeanDifferencePercentage;
}

__host__ void allocateDeviceMemory(int rows, int columns)
{

    //Allocate device constant symbols for rows and columns
    hipMemcpyToSymbol(HIP_SYMBOL(d_rows), &rows, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_columns), &columns, sizeof(int), 0, hipMemcpyHostToDevice);
    int max_pixels = rows * columns;
    hipMemcpyToSymbol(HIP_SYMBOL(d_max_pixels), &max_pixels, sizeof(int), 0, hipMemcpyHostToDevice); // Had to add this as the size of shared variable in kerel can only be specified using a constant and not pdt of 2 constants
}

__host__ void executeKernel(uchar *r, uchar *g, uchar *b, int rows, int columns, int threadsPerBlock)
{
    cout << "Executing kernel\n";
    //Launch the convert CUDA Kernel
    int blocksPerGrid = (rows * columns) / threadsPerBlock;
 
    applySimpleLinearBlurFilter<<<blocksPerGrid, threadsPerBlock>>>(r, g, b); // 1d kernel
    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Reset the device and exit
__host__ void cleanUpDevice()
{
    cout << "Cleaning CUDA device\n";
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipError_t err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__host__ std::tuple<std::string, std::string, std::string, int> parseCommandLineArguments(int argc, char *argv[])
{
    cout << "Parsing CLI arguments\n";
    int threadsPerBlock = 256;
    std::string inputImage = "sloth.png";
    std::string outputImage = "grey-sloth.png";
    std::string currentPartId = "test";

    for (int i = 1; i < argc; i++)
    {
        std::string option(argv[i]);
        i++;
        std::string value(argv[i]);
        if (option.compare("-i") == 0)
        {
            inputImage = value;
        }
        else if (option.compare("-o") == 0)
        {
            outputImage = value;
        }
        else if (option.compare("-t") == 0)
        {
            threadsPerBlock = atoi(value.c_str());
        }
        else if (option.compare("-p") == 0)
        {
            currentPartId = value;
        }
    }
    cout << "inputImage: " << inputImage << " outputImage: " << outputImage << " currentPartId: " << currentPartId << " threadsPerBlock: " << threadsPerBlock << "\n";
    return {inputImage, outputImage, currentPartId, threadsPerBlock};
}

__host__ std::tuple<int, int, uchar *, uchar *, uchar *> readImageFromFile(std::string inputFile)
{
    cout << "Reading Image From File\n";
    Mat img = imread(inputFile, IMREAD_COLOR);
    
    const int rows = img.rows;
    const int columns = img.cols;
    size_t size = sizeof(uchar) * rows * columns;

    cout << "Rows: " << rows << " Columns: " << columns << "\n";

    uchar *r, *g, *b;
    hipMallocManaged(&r, size);
    hipMallocManaged(&g, size);
    hipMallocManaged(&b, size);
    
    for(int y = 0; y < rows; ++y)
    {
        for(int x = 0; x < columns; ++x)
        {
            Vec3b rgb = img.at<Vec3b>(y, x);
            r[y*rows+x] = rgb.val[0];
            g[y*rows+x]= rgb.val[1];
            b[y*rows+x] = rgb.val[2];
        }
    }

    return {rows, columns, r, g, b};
}

__host__ std::tuple<uchar *, uchar *, uchar *>applyBlurKernel(std::string inputImage)
{
    cout << "CPU applying kernel\n";
    Mat img = imread(inputImage, IMREAD_COLOR);
    const int rows = img.rows;
    const int columns = img.cols;

    uchar *r = (uchar *)malloc(sizeof(uchar) * rows * columns);
    uchar *g = (uchar *)malloc(sizeof(uchar) * rows * columns);
    uchar *b = (uchar *)malloc(sizeof(uchar) * rows * columns);

    for(int y = 0; y < rows; ++y)
    {
        for(int x = 1; x < columns-1; ++x)
        {
            Vec3b rgb0 = img.at<Vec3b>(y, x-1);
            Vec3b rgb1 = img.at<Vec3b>(y, x);
            Vec3b rgb2 = img.at<Vec3b>(y, x+1);
            b[y*rows+x] = (rgb0[0] + rgb1[0] + rgb2[0])/3;
            g[y*rows+x] = (rgb0[1] + rgb1[1] + rgb2[1])/3;
            r[y*rows+x] = (rgb0[2] + rgb1[2] + rgb2[2])/3;
        }
    }

    return {r, g, b};
}

int main(int argc, char *argv[])
{
    std::tuple<std::string, std::string, std::string, int> parsedCommandLineArgsTuple = parseCommandLineArguments(argc, argv);
    std::string inputImage = get<0>(parsedCommandLineArgsTuple);
    std::string outputImage = get<1>(parsedCommandLineArgsTuple);
    std::string currentPartId = get<2>(parsedCommandLineArgsTuple);
    int threadsPerBlock = get<3>(parsedCommandLineArgsTuple);
    try 
    {
        auto[rows, columns, r, g, b] = readImageFromFile(inputImage);

        executeKernel(r, g, b, rows, columns, threadsPerBlock);

        Mat colorImage(rows, columns, CV_8UC3);
        vector<int> compression_params;
        compression_params.push_back(IMWRITE_PNG_COMPRESSION);
        compression_params.push_back(9);

        for(int y = 0; y < rows; ++y)
        {
            for(int x = 0; x < columns; ++x)
            {
                colorImage.at<Vec3b>(y,x) = Vec3b(b[y*rows+x], g[y*rows+x], r[y*rows+x]);
            }
        }

        imwrite(outputImage, colorImage, compression_params);

        auto[test_r, test_g, test_b] = applyBlurKernel(inputImage);
        
        float scaledMeanDifferencePercentage = compareColorImages(r, g, b, test_r, test_g, test_b, rows, columns) * 100;
        cout << "Mean difference percentage: " << scaledMeanDifferencePercentage << "\n";

        cleanUpDevice();
    }
    catch (Exception &error_)
    {
        cout << "Caught exception: " << error_.what() << endl;
        return 1;
    }
    return 0;
}