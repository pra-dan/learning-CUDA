#include "hip/hip_runtime.h"
#include "memory_analysis.h"

__global__ void globalMemorySearch(int *input, int totalFound, int numElements, int numThreads)
{
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadId < numThreads)
    {
        // Create a for loop that handles the fact that each thread needs to search for n values in input
            // Increment the input array at index i by 1, update in memory prior to testing if it is equal to constant_search_value
        
        int span = numElements / numThreads;
        int start = threadId * span;
        // So e.g if numElements=100, and numThreads = 4, i.e every thread searches in 25 values, 
        // thread 0 will start = 0*25 and end = 0 + 25
        // thread 1 will start = 1*25 and end = 25 + 25... etc
        int end = start + span;

        for (int i=start; i<end; i++){
            input[i]++;
            if(input[i] == constant_search_value)
                input[i] = 1;
        }
    }
}

// This one is definitely forced and seems dumb tbh!!!
__global__ void sharedMemorySearch(int *input, int totalFound, int numElements, int numThreads)
{
    extern __shared__ int sharedInput[];
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;

    int span = numElements / numThreads;
    int start = threadId * span;

    // Load to shared memory
    for (int i = threadIdx.x; i < numElements; i += blockDim.x)
        sharedInput[i] = input[i];
    __syncthreads();

    if (threadId < numThreads)
    {
        for (int i = start; i < start + span; i++) {
            sharedInput[i] += 1;
            if (sharedInput[i] == constant_search_value) {
                sharedInput[i] = 1;
            }
        }
    }
    __syncthreads();

    for (int i = threadIdx.x; i < numElements; i += blockDim.x)
        input[i] = sharedInput[i];
}

// The unique part here is that input is also passed as a constant
__global__ void constantMemorySearch(int totalFound)
{
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadId < constant_num_threads)
    {
        // Create a for loop that handles the fact that each thread needs to search for n values in input
            // Increment the input array at index i by 1, update in memory prior to testing if it is equal to constant_search_value

        int start = threadId * constant_thread_span;
        int end = start + constant_thread_span;
        for (int i=start; i<end; i++){
            // Ignoring increment step as we can't modify a constant, rather do it with totalFound
            totalFound = 0;
            if(constant_input[i] == constant_search_value)
                totalFound += 1;
        }
    }
}

__global__ void registerMemorySearch(int *input, int totalFound, int numElements, int numThreads)
{
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadId < numThreads)
    {
        // Create a for loop that handles the fact that each thread needs to search for n values in input
            // Increment the input array at index i by 1, update in memory prior to testing if it is equal to constant_search_value
        
        int span = numElements / numThreads;
        int start = threadId * span;
        // So e.g if numElements=100, and numThreads = 4, i.e every thread searches in 25 values, 
        // thread 0 will start = 0*25 and end = 0 + 25
        // thread 1 will start = 1*25 and end = 25 + 25... etc
        int end = start + span;

        for (int i=start; i<end; i++){
            int val = input[i];
            val++; 
            if(val == constant_search_value)
                val = 1;
            input[i] = val; //copy back from register to global memory
        }
    }
}

// This will generate an array of size numElements of random integers from 0 to 255 in pageable host memory
__host__ int * allocatePageableRandomHostMemory(int numElements)
{
    srand(time(0));
    size_t size = numElements * sizeof(int);

    // Allocate the host pinned memory input pointer B
    int *data;
    hipHostAlloc((void**)&data, size, hipHostMallocDefault);

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        data[i] = rand() % 255;
    }

    return data;
}

__host__ int * allocateDeviceMemory(int numElements)
{
    size_t size = numElements * sizeof(int);

    int *d_input = NULL;
    hipError_t err = hipMalloc(&d_input, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_input (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return d_input;
}

__host__ void copyFromHostToDevice(std::string kernelType, int *input, int numElements, int numThreads, int *d_input)
{
    size_t size = numElements * sizeof(int);

    if(!strcmp(kernelType.c_str(), "constant"))
    {
        // Copy input, numElements, threadSpan, and numThreads to constant memory
        int threadSpan = numElements / numThreads;
        int search_value = 100; // I defined it as the author missed it
        hipMemcpyToSymbol(HIP_SYMBOL(constant_input), input, size, 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(constant_search_value), &(search_value), sizeof(int), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(constant_num_elements), &(numElements), sizeof(int), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(constant_num_threads), &(numThreads), sizeof(int), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(constant_thread_span), &(threadSpan), sizeof(int), 0, hipMemcpyHostToDevice);
    } else 
    {
        // Copy input to d_input 
        hipError_t err = hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy array input from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }
}

__host__ void executeKernel(int *d_input, int numElements, int threadsPerBlock, std::string kernelType)
{
    int totalFound = 0;
    // Launch the search CUDA Kernel
    if (!strcmp(kernelType.c_str(), "global"))
    {
        globalMemorySearch<<<1,threadsPerBlock>>>(d_input, totalFound, numElements, threadsPerBlock); // you will need to fill in function arguments appropriately
    } else if (!strcmp(kernelType.c_str(), "constant"))
    {
        constantMemorySearch<<<1,threadsPerBlock>>>(totalFound);  // you will need to fill in function arguments appropriately
    } else if (!strcmp(kernelType.c_str(), "shared"))
    {
        unsigned int_array_size = numElements * sizeof(int);
        sharedMemorySearch<<<1,threadsPerBlock, int_array_size>>>(d_input, totalFound, numElements, threadsPerBlock); // you will need to fill in function arguments appropriately
    } else {
        registerMemorySearch<<<1,threadsPerBlock>>>(d_input, totalFound, numElements, threadsPerBlock);  // you will need to fill in function arguments appropriately
    }
    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch %s kernel (error code %s)!\n", kernelType.c_str(), hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();
}

// Free device global memory
__host__ void deallocateMemory(int *d_input)
{

    hipError_t err = hipFree(d_input);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_input (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

}

// Reset the device and exit
__host__ void cleanUpDevice()
{
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipError_t err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


__host__ std::tuple<int, std::string, int, std::string> parseCommandLineArguments(int argc, char *argv[])
{
    int elementsPerThread = 2;
    int threadsPerBlock = 256;
    std::string currentPartId = "test";
    std::string kernelType = "global";

    for(int i = 1; i < argc; i++)
    {
        std::string option(argv[i]);
        i++;
        std::string value(argv[i]);
        if(option.compare("-t") == 0) 
        {
            threadsPerBlock = atoi(value.c_str());
        }
        else if(option.compare("-m") == 0) 
        {
            elementsPerThread = atoi(value.c_str());
        }
        else if(option.compare("-p") == 0) 
        {
            currentPartId = value;
        }
        else if(option.compare("-k") == 0) 
        {
            kernelType = value;
        }
    }

    return {elementsPerThread, currentPartId, threadsPerBlock, kernelType};
}

__host__ int * setUpInput(int numElements)
{
    srand(time(0));
    int *input;

    int searchValue = rand() % 255;
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(constant_search_value), &searchValue, sizeof(int), 0, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy constant int d_v from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    input = allocatePageableRandomHostMemory(numElements);

    return input;
}

/*
 * Host main routine
 * -m elementsPerThread - the number of elements that a thread will search for a random value in
 * -p currentPartId - the Coursera Part ID
 * -t threadsPerBlock - the number of threads to schedule for concurrent processing
 * -k the kernel type - global, constant, shared, register
 */
int main(int argc, char *argv[])
{
    auto[elementsPerThread, currentPartId, threadsPerBlock, kernelType] = parseCommandLineArguments(argc, argv);

    int numElements = elementsPerThread * threadsPerBlock;

    int *input = setUpInput(numElements);
    int *d_input = allocateDeviceMemory(numElements);

    copyFromHostToDevice(kernelType, input, numElements, threadsPerBlock, d_input);

    // Start time including kernel processing time
    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventRecord(start,0);

    executeKernel(d_input, numElements, threadsPerBlock, kernelType);

    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start,stop);

    deallocateMemory(d_input);
    cleanUpDevice();
    
    ofstream outputfile;
    outputfile.open ("output.csv", std::ios_base::app);
    outputfile << currentPartId.c_str() << "," << kernelType.c_str() << "," << threadsPerBlock << "," << elementsPerThread << "," << elapsedTime << "\n";
    outputfile.close();

    return 0;
}